#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

// function to integrate
#define F(x) (x*x)

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); exit(-1);} 

const long blocks_per_grid = 128;
const long threads_per_block = 128;  // Must be a power of 2 for reduction 

__device__ double rn(unsigned long * seed)
{
  double ret;
  unsigned long n1;
  unsigned long a = 16807;
  unsigned long m = 2147483647;
  n1 = ( a * (*seed) ) % m;
  *seed = n1;
  ret = (double) n1 / m;
  return ret;
}

__global__ void lookup(double *F_vals, long F_len, double interval, 
     long total_lookups, double *sums) {

  __shared__ double sum_cache[threads_per_block];
  long i,j,k;
  double x, f;
  unsigned long seed;

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int cache_id = threadIdx.x;

  seed = 10000*threadIdx.x + 10* blockIdx.x + threadIdx.x;

  for (i=thread_id; i < total_lookups; i += gridDim.x*blockDim.x) {

    // Randomly sample a continous value for x
    x = (double) rn(&seed);

    // Find the indices that bound x on the grid
    j = x / interval;
    k = j+1;

    // Calculate interpolation factor
    f = (k*interval - x) / (k*interval - j*interval);

    // Interpolate and accumulate result
    sum_cache[cache_id] += F_vals[j+1] - f * (F_vals[j+1] - F_vals[j]);
  }

  __syncthreads();

  // Reduction
  for (i=blockDim.x/2; i != 0; i /= 2) {
    if (cache_id < i)
      sum_cache[cache_id] += sum_cache[cache_id + i];
    __syncthreads();
  }

  if (cache_id == 0) 
    sums[blockIdx.x] = sum_cache[0];

}



int main(int argc, char* argv[]) {

  // number of lookups
  long n_lookups = (argc < 2) ? 10000000 : atol(argv[1]);  
  // number of gridpoints
  long F_len  = (argc < 3) ? 250000000 : atol(argv[2]);    
  // Discrete values for F(x)
  double *F_vals, *dev_F_vals;
  // interval for linearly-spaced grid
  double interval = (double) 1 / (F_len - 1);
  // Sum of random lookups on F_vals
  double sum = 0;
  long i;

  double *sums, *dev_sums;
  // struct timeval start, end; // start and end times
  // double wall_time;  // wall_time elapsed

  printf("Running %0.2e lookups with %0.2e gridpoints in a %0.2f MB array...\n", 
      (double) n_lookups, (double) F_len, (double) F_len*sizeof(double)/1e6);


  sums = (double *) malloc( blocks_per_grid*sizeof(double) );
  F_vals = (double *) malloc(F_len*sizeof(double));


  // Populate values for F(x) on grid
  for (i=0; i<F_len; i++) {
    F_vals[i] = F(i*interval);
  }

  CUDA_CALL( hipMalloc( (void**)&dev_sums, blocks_per_grid*sizeof(double) ) );
  CUDA_CALL( hipMemset( (void*) dev_sums, 0, blocks_per_grid*sizeof(double) ) );


  CUDA_CALL( hipMalloc( (void**)&dev_F_vals, F_len*sizeof(double)) );
  CUDA_CALL( hipMemcpy( dev_F_vals, F_vals, F_len*sizeof(double), hipMemcpyHostToDevice ) );

  lookup<<<blocks_per_grid,threads_per_block>>>(dev_F_vals, F_len, interval, n_lookups, dev_sums);

  CUDA_CALL( hipMemcpy( sums, dev_sums, threads_per_block*sizeof(double), hipMemcpyDeviceToHost ));


  // gettimeofday(&start, NULL);


  // gettimeofday(&end, NULL);

  // wall_time = (end.tv_sec - start.tv_sec)*1000000 + (end.tv_usec - start.tv_usec);

  for (i=0; i<blocks_per_grid; i++) {
    sum += sums[i];
  }

  printf("Result: %0.6f\n", sum / n_lookups);
  // printf("Time:   %0.2e s\n", wall_time);
  // printf("Rate:   %0.2e lookups/s\n", n_lookups / wall_time);

  return 0;
}
