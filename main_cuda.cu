#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

// function to integrate
#define F(x) (x*x)

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); exit(-1);} 

const long blocks_per_grid = 64;
const long threads_per_block = 128;  // Must be a power of 2 for reduction 

__device__ double rn(unsigned long * seed)
{
  double ret;
  unsigned long n1;
  unsigned long a = 16807;
  unsigned long m = 2147483647;
  n1 = ( a * (*seed) ) % m;
  *seed = n1;
  ret = (double) n1 / m;
  return ret;
}

__global__ void lookup(double *F_vals, long F_len, double interval, 
    long total_lookups, double *sums) {

  // A per-block cache.  Each thread i writes to sum_cache[i]
  __shared__ double sum_cache[threads_per_block];

  long i,j,k;
  double x, f;
  unsigned long seed;

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int cache_id = threadIdx.x;

  seed = 10000*threadIdx.x + 10* blockIdx.x + threadIdx.x;

  for (i=thread_id; i < total_lookups; i += gridDim.x*blockDim.x) {

    // Randomly sample a continous value for x
    x = (double) rn(&seed);

    // Find the indices that bound x on the grid
    j = x / interval;
    k = j+1;

    // Calculate interpolation factor
    f = (k*interval - x) / (k*interval - j*interval);

    // Interpolate and accumulate result
    sum_cache[cache_id] += F_vals[j+1] - f * (F_vals[j+1] - F_vals[j]);
  }

  __syncthreads();

  // Naive reduction
  for (i=blockDim.x/2; i != 0; i /= 2) {
    if (cache_id < i)
      sum_cache[cache_id] += sum_cache[cache_id + i];
    __syncthreads();
  }
  if (cache_id == 0) 
    sums[blockIdx.x] = sum_cache[0];

}



int main(int argc, char* argv[]) {

  // number of lookups
  long n_lookups = (argc < 2) ? 10000000 : atol(argv[1]);  
  // number of gridpoints
  long F_len  = (argc < 3) ? 250000000 : atol(argv[2]);    
  // Discrete values for F(x)
  double *F_vals, *dev_F_vals;
  // interval for linearly-spaced grid
  double interval = (double) 1 / (F_len - 1);
  // Sum of random lookups on F_vals
  double sum = 0;
  // Vectors for sums of F(x_i).  Dimensions will be sums[0:blocks_per_grid].
  // Each block j will reduce is results to sum[i].
  double *sums, *dev_sums;
  // Timing
  hipEvent_t start, stop;
  float elapsed_time;
  // Loop control
  long i;

  printf("Running %0.2e lookups with %0.2e gridpoints in a %0.2f MB array...\n", 
      (double) n_lookups, (double) F_len, (double) F_len*sizeof(double)/1e6);

  CUDA_CALL( hipEventCreate( &start ) );
  CUDA_CALL( hipEventCreate( &stop ) );

  sums = (double *) malloc( blocks_per_grid*sizeof(double) );
  F_vals = (double *) malloc(F_len*sizeof(double));

  // Populate values for F(x) on grid
  for (i=0; i<F_len; i++) {
    F_vals[i] = F(i*interval);
  }

  // Malloc and set dev_sums
  CUDA_CALL( hipMalloc( (void**)&dev_sums, blocks_per_grid*sizeof(double) ) );
  CUDA_CALL( hipMemset( (void*) dev_sums, 0, blocks_per_grid*sizeof(double) ) );

  // Malloc and copyto dev_F_vals
  CUDA_CALL( hipMalloc( (void**)&dev_F_vals, F_len*sizeof(double)) );
  CUDA_CALL( hipMemcpy( dev_F_vals, F_vals, F_len*sizeof(double), hipMemcpyHostToDevice ) );

  CUDA_CALL( hipEventRecord( start, 0 ) );

  lookup<<<blocks_per_grid,threads_per_block>>>(dev_F_vals, F_len, interval, n_lookups, dev_sums);

  CUDA_CALL( hipEventRecord( stop, 0 ) );
  CUDA_CALL( hipEventSynchronize( stop ) );

  // Copy dev_sums to sums
  CUDA_CALL( hipMemcpy( sums, dev_sums, blocks_per_grid*sizeof(double), hipMemcpyDeviceToHost ));

  // Get cumulative sum
  for (i=0; i<blocks_per_grid; i++) {
    sum += sums[i];
  }

  // Get timings
  CUDA_CALL( hipEventElapsedTime( &elapsed_time, start, stop ) );
  printf("Result: %0.6f\n", sum / n_lookups);
  printf("Time:   %0.2e s\n", elapsed_time);
  printf("Rate:   %0.2e lookups/s\n", n_lookups / elapsed_time);

  // Cleanup
  CUDA_CALL( hipEventDestroy( start ) );
  CUDA_CALL( hipEventDestroy( stop ) );
  CUDA_CALL( hipFree( dev_F_vals ) );
  CUDA_CALL( hipFree( dev_sums ) );
  free(F_vals);
  free(sums);

  return 0;
}
