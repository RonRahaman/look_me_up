#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

// function to integrate
#define F(x) (x*x)

__device__ double rn(unsigned long * seed)
{
  double ret;
  unsigned long n1;
  unsigned long a = 16807;
  unsigned long m = 2147483647;
  n1 = ( a * (*seed) ) % m;
  *seed = n1;
  ret = (double) n1 / m;
  return ret;
}

__global__ void lookup(double *F_vals, long n_grid, double interval, 
    double *sums, int n_sums, long lookups_per_thread, long total_lookups) {

  long i,j,k;
  double x, f;
  unsigned long seed;

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < n_sums) {

    seed = 10000*threadIdx.x + 10* blockIdx.x + threadIdx.x;

    for (i=0; i<n_lookups; i++) {

      // Randomly sample a continous value for x
      x = (double) rn(&seed);

      // Find the indices that bound x on the grid
      j = x / interval;
      k = j+1;

      // Calculate interpolation factor
      f = (k*interval - x) / (k*interval - j*interval);

      // Interpolate and accumulate result
      sums[tid] += F_vals[j+1] - f * (F_vals[j+1] - F_vals[j]);
    }

}



int main(int argc, char* argv[]) {

  // number of lookups
  long total_lookups = (argc < 2) ? 10000000 : atol(argv[1]);  
  // number of gridpoints
  long n_grid  = (argc < 3) ? 250000000 : atol(argv[2]);    
  // Discrete values for F(x)
  double * F_vals = (double *) malloc(n_grid*sizeof(double));
  // interval for linearly-spaced grid
  double interval = (double) 1 / (n_grid - 1);
  // Sum of random lookups on F_vals
  double sum = 0;

  int threads_per_block = 500;
  long lookups_per_thread = 10000;
  int n_blocks = (total_lookups + threads_per_block*lookups_per_thread - 1) /
    (threads_per_block*lookups_per_thread);
  int n_threads

  dim3 dim_block(threads_per_block, 1, 1);
  dim3 dim_grid(n_blocks, 1, 1)

  // struct timeval start, end; // start and end times
  // double wall_time;  // wall_time elapsed

  printf("Running %0.2e lookups with %0.2e gridpoints in a %0.2f MB array...\n", 
      (double) n_lookups, (double) n_grid, (double) n_grid*sizeof(double)/1e6);

  // Populate values for F(x) on grid
  for (i=0; i<n_grid; i++) {
    F_vals[i] = F(i*interval);
  }

  // gettimeofday(&start, NULL);


  // gettimeofday(&end, NULL);

  // wall_time = (end.tv_sec - start.tv_sec)*1000000 + (end.tv_usec - start.tv_usec);

  printf("Result: %0.6f\n", sum / n_lookups);
  // printf("Time:   %0.2e s\n", wall_time);
  // printf("Rate:   %0.2e lookups/s\n", n_lookups / wall_time);

  return 0;
}
